#include "hip/hip_runtime.h"
#include "kmeans.h"


__device__ float square_l2_distance(float x1,float y1,float x2,float y2){
    return (x1-x2)*(x1-x2) + (y1-y2)*(y1-y2);
}

// __restrict__ is pointer aliasing, where the same memory location can be accessed using different names 
__global__ void assign_clusters(const float* __restrict__ data_x,const float* __restrict__ data_y,int data_size,const float* __restrict__ means_x,const float* __restrict__ means_y,
                                float* __restrict__ new_sums_x,float* __restrict__ new_sums_y,int k,int* __restrict__ counts){
    const int index = blockIdx.x * blockDim.x + threadIdx.x;

    if(index>=data_size) return;

    float best_dist = FLT_MAX;
    int best_cluster = 0;
    for(int cluster=0;cluster<k;++cluster){
        const float dist = square_l2_distance(data_x[index],data_y[index],means_x[cluster],means_y[cluster]);
        if(dist<best_dist){
            best_dist = dist;
            best_cluster = cluster;
        }
    }
    //atomic operation (read-->modify-->overwrite)
    atomicAdd(&new_sums_x[best_cluster],data_x[index]);
    atomicAdd(&new_sums_y[best_cluster],data_y[index]);
    atomicAdd(&counts[best_cluster],1);
}

__global__ void compute_new_means(float* __restrict__ means_x,float* __restrict__ means_y,const float* __restrict__ new_sum_x,const float* __restrict__ new_sum_y,const int* __restrict__ counts) {
    const int cluster = threadIdx.x;
    const int count = max(1,counts[cluster]);
    means_x[cluster] = new_sum_x[cluster]/count;
    means_y[cluster] = new_sum_y[cluster]/count;
}

int main(int argc, const char* argv[]) {
    std::vector<float> h_x;
    std::vector<float> h_y;

    std::string line;
    std::ifstream infile("x.txt");
    if(infile.is_open()){
        while(std::getline(infile,line)){
            std::istringstream stream(line);
            float x1,y1;
            stream>>x1>>y1;
            h_x.push_back(x1);
            h_y.push_back(y1);
        }
    }

    infile.close();

    const size_t number_of_elements = h_x.size();

    Data d_data(number_of_elements, h_x, h_y);

    const size_t k =2, number_of_iterations = 50;

    // Random shuffle the data and pick the first
    // k points (i.e. k random points).
    std::random_device seed;
    std::mt19937 rng(seed());
    std::shuffle(h_x.begin(), h_x.end(), rng);
    std::shuffle(h_y.begin(), h_y.end(), rng);
    Data d_means(k, h_x, h_y);

    Data d_sums(k);

    int* d_counts;
    hipMalloc(&d_counts, k * sizeof(int));
    hipMemset(d_counts, 0, k * sizeof(int));

    const int threads = 1024;
    const int blocks = (number_of_elements + threads - 1) / threads;

    for (size_t iteration = 0; iteration < number_of_iterations; ++iteration) {
        hipMemset(d_counts, 0, k * sizeof(int));
        d_sums.clear();

        assign_clusters<<<blocks, threads>>>(d_data.d_x,d_data.d_y,d_data.size,d_means.d_x,d_means.d_y,d_sums.d_x,d_sums.d_y,k,d_counts);
        hipDeviceSynchronize();

        compute_new_means<<<1, k>>>(d_means.d_x,d_means.d_y,d_sums.d_x,d_sums.d_y,d_counts);
        hipDeviceSynchronize();
    }

    std::vector<float> h_mean_x(k);
    std::vector<float> h_mean_y(k);
    hipMemcpy(h_mean_x.data(),d_means.d_x,k*sizeof(float),hipMemcpyDeviceToHost);
    hipMemcpy(h_mean_y.data(),d_means.d_y,k*sizeof(float),hipMemcpyDeviceToHost);

    for(int i=0;i<k;++i){
        std::cout << h_mean_x[i] << " " << h_mean_y[i] << std::endl;
    }

}